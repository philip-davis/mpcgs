#include "hip/hip_runtime.h"
#include "tree.cuh"

extern "C" {

#ifndef MPCGS_NOGPU

#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "phylip.h"
#include "mpcgs-gpu.h"
#include "tree.h"

__constant__ float lfreq[MPCGS_NUM_FREQ_TERM];

__device__ static float sum_calc_lposterior_gpu(struct gtree_summary *sum,
                                                float theta)
{

    int i;
    int lineages;
    float exp1, coeff;

    exp1 = 0;
    for (i = 0; i < sum->nintervals; i++) {
        lineages = i + 2;
        exp1 += -(float)(lineages * (lineages - 1)) * sum->intervals[i];
    }
    coeff = (float)(lineages - 1) * logf(2.0 / theta);

    return (coeff + (exp1 / theta));
}

__global__ static void set_base_lposteriors(struct gtree_summary_set *sum_set,
                                            unsigned int nsummaries,
                                            float theta)
{

    unsigned int sum_idx;
    struct gtree_summary *sum;

    sum_idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (sum_idx < nsummaries) {
        sum = &sum_set->summaries[sum_idx];
        sum->ldrv_posterior = sum_calc_lposterior_gpu(sum, theta);
    }
}

//TODO: rename to distinguish theta likelihood from tree likelihood
__global__ static void set_llhood_comps(struct gtree_summary_set *sum_set,
                                        unsigned int nsummaries,
                                        float theta)
{

    unsigned sum_idx, delta;
    extern __shared__ float warp_normal[];
    struct gtree_summary *sum;
    float lcomp, other_lcomp;
    int i;

    sum_idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (sum_idx < nsummaries) {
        sum = &sum_set->summaries[sum_idx];
        lcomp = sum_calc_lposterior_gpu(sum, theta) - sum->ldrv_posterior;
        sum->ltmp_lkhood_comp = lcomp;
    } else {
        lcomp = -FLT_MAX;
    }
    __syncthreads();
    for (delta = (WARPSZ / 2); delta >= 1; delta /= 2) {
        other_lcomp = __shfl_down(lcomp, delta);
        lcomp = fmaxf(lcomp, other_lcomp);
    }
    if ((sum_idx % WARPSZ) == 0) {
        warp_normal[threadIdx.x / WARPSZ] = lcomp;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 0; i < DEF_BLKSZ / WARPSZ; i++) {
            lcomp = fmaxf(lcomp, warp_normal[i]);
        }
        sum_set->block_scratch[blockIdx.x] = lcomp;
    }
}

__global__ static void sum_comp_fracs(struct gtree_summary_set *sum_set,
                                      unsigned int nsummaries,
                                      float normal)
{
    unsigned sum_idx, delta;
    extern __shared__ float warp_sum[];
    struct gtree_summary *sum;
    float comp, other_comp, sum_comp;
    int i;

    sum_idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (sum_idx < nsummaries) {
        sum = &sum_set->summaries[sum_idx];
        comp = expf(sum->ltmp_lkhood_comp - normal);
    } else {
        comp = 0;
    }
    __syncthreads();

    for (delta = (WARPSZ / 2); delta >= 1; delta /= 2) {
        other_comp = __shfl_down(comp, delta);
        comp += other_comp;
    }
    if ((sum_idx % WARPSZ) == 0) {
        warp_sum[threadIdx.x / WARPSZ] = comp;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        sum_comp = 0;
        for (i = 0; i < DEF_BLKSZ / WARPSZ; i++) {
            sum_comp += warp_sum[i];
        }
        sum_set->block_scratch[blockIdx.x] = sum_comp;
    }
}

float gtree_summary_set_llkhood_gpu(struct gtree_summary_set *sum_set,
                                    float theta)
{

    size_t num_block, shared_size;
    float normal, lkhood;
    unsigned i;

    num_block = (size_t)ceil((float)sum_set->nsummaries / (float)DEF_BLKSZ);
    shared_size = (size_t)ceil(DEF_BLKSZ / WARPSZ) * sizeof(float);

    set_llhood_comps<<<num_block, DEF_BLKSZ, shared_size>>>(
      sum_set, sum_set->nsummaries, theta);
    hipDeviceSynchronize();

    normal = -FLT_MAX;
    for (i = 0; i < num_block; i++) {
        normal = fmaxf(normal, sum_set->block_scratch[i]);
    }

    sum_comp_fracs<<<num_block, DEF_BLKSZ, shared_size>>>(
      sum_set, sum_set->nsummaries, normal);
    hipDeviceSynchronize();

    lkhood = 0;
    for (i = 0; i < num_block; i++) {
        lkhood += sum_set->block_scratch[i];
    }

    return (logf(lkhood) + normal);
}

void gtree_summary_set_base_lposteriors_gpu(struct gtree_summary_set *sum_set,
                                            float drv_theta)
{

    size_t num_block;

    if (!sum_set) {
        // TODO: handle error
    }

    if (drv_theta <= 0.0) {
        // TODO: handle error
    }

    num_block = (size_t)ceil((float)sum_set->nsummaries / (float)DEF_BLKSZ);

    set_base_lposteriors<<<num_block, DEF_BLKSZ>>>(
      sum_set, sum_set->nsummaries, drv_theta);
    hipDeviceSynchronize();
}

void gtree_summary_set_create_gpu(struct gtree_summary_set **sum_set,
                                  size_t count,
                                  size_t nintervals)
{

    int i;
    struct gtree_summary *summary;
    size_t num_block;

    if (!sum_set) {
        // TODO: handle error
    }

    hipMallocManaged(sum_set, sizeof(**sum_set));

    (*sum_set)->nsummaries = 0;
    (*sum_set)->szintervals = count;
    hipMallocManaged(&(*sum_set)->summaries,
                      count * sizeof(*((*sum_set)->summaries)));
    if (!(*sum_set)->summaries) {
        // TODO: handle error
    }

    summary = (*sum_set)->summaries;
    for (i = 0; i < count; i++) {
        summary->nintervals = nintervals;
        hipMallocManaged(&summary->intervals,
                          nintervals * sizeof(*(summary->intervals)));
        if (!summary->intervals) {
            // TODO: handle error
        }
        summary++;
    }

    num_block = (size_t)ceil((float)count / (float)DEF_BLKSZ);
    hipMallocManaged(&((*sum_set)->block_scratch), num_block * sizeof(float));
    hipDeviceSynchronize();

}

__device__ static void gnode_get_llhood_lcomps_gpu(struct gene_node *gnode,
                                    float *cllike,
                                    float *lcomps)
{

    float sumAllfact, lsumAll, sumPur, lsumPur, sumPyr, lsumPyr;
    float normal, comp;
    int i;

    /************************************************************
     * The following code adapted from LAMARC, (c) 2002
     * Peter Beerli, Mary Kuhner, Jon  Yamato and Joseph Felsenstein
     * TODO: license ref?
     ***********************************************************/
    normal = -FLT_MAX;
    for (i = 0; i < NUM_BASE; i++) {
        normal = fmaxf(normal, (lfreq[i] + cllike[i]));
    }
    sumAllfact = 0;
    for (i = 0; i < NUM_BASE; i++) {
        if (cllike[i] > -FLT_MAX) {
            sumAllfact += exp((lfreq[i] + cllike[i]) - normal);
        }
    }
    lsumAll = gnode->lexpA + log(sumAllfact) + normal;

    normal =
      fmaxf((lfreq[FREQ_AR] + cllike[DNA_A]), (lfreq[FREQ_GR] + cllike[DNA_G]));
    sumPur = 0;
    if (cllike[DNA_A] > -FLT_MAX) {
        sumPur += exp((lfreq[FREQ_AR] + cllike[DNA_A]) - normal);
    }
    if (cllike[DNA_G] > -FLT_MAX) {
        sumPur += exp((lfreq[FREQ_GR] + cllike[DNA_G]) - normal);
    }

    if (sumPur > 0) {
        lsumPur = log(sumPur) + normal;
    } else {
        lsumPur = -FLT_MAX;
    }

    normal =
      fmaxf((lfreq[FREQ_CY] + cllike[DNA_C]), (lfreq[FREQ_TY] + cllike[DNA_T]));
    sumPyr = 0;
    if (cllike[DNA_C] > -FLT_MAX) {
        sumPyr += exp((lfreq[FREQ_CY] + cllike[DNA_C]) - normal);
    }
    if (cllike[DNA_T] > -FLT_MAX) {
        sumPyr += exp((lfreq[FREQ_TY] + cllike[DNA_T]) - normal);
    }

    if (sumPyr > 0) {
        lsumPyr = log(sumPyr) + normal;
    } else {
        lsumPyr = -FLT_MAX;
    }

    for (i = 0; i < NUM_BASE; i++) {
        // TODO: place these components into an array rather than recalc?

        normal = fmaxf(lsumAll, (gnode->lexpB + cllike[i]));
        normal = fmaxf(
          normal,
          (gnode->lexpC + ((i == DNA_A || i == DNA_G) ? lsumPur : lsumPyr)));

        comp = exp(lsumAll - normal);
        comp += exp((gnode->lexpB + cllike[i]) - normal);
        comp += exp(
          (gnode->lexpC + ((i == DNA_A || i == DNA_G) ? lsumPur : lsumPyr)) -
          normal);
        lcomps[i] = log(comp) + normal;
    }
    /***********************************************************/
}

__global__ void gtree_compute_llhood(struct gene_tree *gtree, unsigned nseq, unsigned seq_len)
{

	extern __shared__ float s[];
	float *warp_sum = s;
	float *llhood_scratch = &warp_sum[(int)ceil((float)blockDim.x/(float)WARPSZ)];
	struct gene_node *node, *child1, *child2;
	float *llhoods, *tip_llhoods, *ch1_llhoods, *ch2_llhoods;
	float ch1_llhood_comp[NUM_BASE], ch2_llhood_comp[NUM_BASE];
	float normal, pos_lhood, llhood, other_llhood, sum_llhood;
	unsigned delta;
	int i, j, node_idx, ch1_idx, ch2_idx, seq_idx, scratch_idx, scratch_idx_ch1, scratch_idx_ch2;

	//Each thread is a base-pair position.
	seq_idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	if(seq_idx < seq_len) {
		//First copy in pre-computed likelihood data from the tips (this will never change over the life of the program.
		for(i = 0; i < nseq; i++) {
			node = &gtree->tips[i];
			node_idx = node->idx;

			scratch_idx = NUM_BASE * ((node_idx * blockDim.x) + threadIdx.x); //try to combine reads as much as possible.

			llhoods = &llhood_scratch[scratch_idx];
			tip_llhoods = &node->tip_llhoods[NUM_BASE * seq_idx];
			for(j = 0 ; j < NUM_BASE; j++) {
				llhoods[j] = tip_llhoods[j];
			}
		}

		//Next walk backwards from the last coalescent event
		for(node = gtree->last; node; node = node->prev) {
			//TODO: see if any of these can be precomputed to save memory accesses
			child1 = node->child1;
			child2 = node->child2;
			node_idx = node->idx;
			ch1_idx = child1->idx;
			ch2_idx = child2->idx;

			scratch_idx = NUM_BASE * ((node_idx * blockDim.x) + threadIdx.x);
			scratch_idx_ch1 = NUM_BASE * ((ch1_idx * blockDim.x) + threadIdx.x);
			scratch_idx_ch2 = NUM_BASE * ((ch2_idx * blockDim.x) + threadIdx.x);

			llhoods = &llhood_scratch[scratch_idx];
			ch1_llhoods = &llhood_scratch[scratch_idx_ch1];
			ch2_llhoods = &llhood_scratch[scratch_idx_ch2];
			gnode_get_llhood_lcomps_gpu(child1, ch1_llhoods, ch1_llhood_comp);
			gnode_get_llhood_lcomps_gpu(child2, ch2_llhoods, ch2_llhood_comp);
			for(i = 0; i < NUM_BASE; i++) {
				llhoods[i] = ch1_llhood_comp[i] + ch2_llhood_comp[i];
			}
		}

		//Now calculate the root likelihood at this base position.
		//after the loop, llhoods should point at the root likelihood components.
		normal = -FLT_MAX;
		for(i = 0; i < NUM_BASE; i++) {
			normal = fmaxf(normal, llhoods[i]);
		}
		pos_lhood = 0;
		for(i = 0; i < NUM_BASE; i++) {
			pos_lhood += expf((llhoods[i] + lfreq[i]) - normal);
		}
		llhood = logf(pos_lhood) + normal;
	} else {
		llhood = 0;
	}
	__syncthreads();

	//Now sum the llhoods across the block
	for (delta = (WARPSZ / 2); delta >= 1; delta /= 2) {
		other_llhood = __shfl_down(llhood, delta);
		llhood += other_llhood;
	}
	if ((seq_idx % WARPSZ) == 0) {
		warp_sum[threadIdx.x / WARPSZ] = llhood;
	}
	__syncthreads();

	if (threadIdx.x == 0) {
		sum_llhood = 0;
		for (i = 0; i < (int)ceil((float)blockDim.x/(float)WARPSZ); i++) {
			sum_llhood += warp_sum[i];
		}
		gtree->block_scratch[blockIdx.x] = sum_llhood;
	}

}

void gtree_set_llhood_gpu(struct gene_tree *gtree)
{

	size_t num_blocks, block_size, shared_size;
	int i = 0;
	float llhood;

	if(!gtree) {
		//TODO: handle error
	}

	block_size = gtree->block_size;
	num_blocks = gtree->num_blocks;
	shared_size = gtree->shared_size;

	gtree_compute_llhood<<<num_blocks, block_size, shared_size>>>
			(gtree, gtree->ntips, gtree->mstab->seq_len);
	hipDeviceSynchronize();

	llhood = 0;
	for(i = 0; i < num_blocks; i++) {
		llhood += gtree->block_scratch[i];
	}

	gtree->llhood = llhood;

}

static void gnode_add_seq(struct gene_node *tip, struct mol_seq *mseq)
{

	size_t tip_llhood_sz;
	float *llhood;
	int i, j;

	if(!tip || !mseq) {
		//TODO: handle error
	}

	if(tip->child1 || tip->child2) {
		//TODO: warn
	}

	tip_llhood_sz = mseq->len * NUM_BASE * sizeof(*tip->tip_llhoods);
	hipMallocManaged(&tip->tip_llhoods, tip_llhood_sz);

	llhood = tip->tip_llhoods;
	for(i = 0; i < mseq->len; i++) {
		for(j = 0; j < NUM_BASE; j++) {
			if(j == mseq->seq[i]) {
				llhood[j] = 0;
			} else {
				llhood[j] = -FLT_MAX;
			}
		}
		llhood += NUM_BASE;
	}

	tip->mseq = mseq;

	hipDeviceSynchronize();

}

void gtree_add_seqs_to_tips_gpu(struct gene_tree *gtree, struct ms_tab *mstab)
{

    int i;
    unsigned int mol_counts[PHY_NUM_MOL_T] = { 0 };
    float freqa, freqg, freqc, freqt; // for readability
    float freqar, freqgr, freqcy, freqty;
    float pur, pyr, ag, ct, m, n, fracchange;
    unsigned int nmol;
    //hipError_t cucode;

    if (!gtree || !mstab) {
        // TODO: handle error
    }

    if (gtree->ntips != mstab->len) {
        // TODO: handle error
    }

    gtree->mstab = mstab;

    for (i = 0; i < mstab->len; i++) {
        gtree->tips[i].mseq = &mstab->mseq[i];
        gnode_add_seq(&(gtree->tips[i]), &(mstab->mseq[i]));
    }
    nmol = get_mol_counts(mstab, mol_counts);
    if (!nmol) {
        // TODO: handle error
    }

    freqa = (float)mol_counts[DNA_A] / (float)nmol;
    freqt = (float)mol_counts[DNA_T] / (float)nmol;
    freqc = (float)mol_counts[DNA_C] / (float)nmol;
    freqg = (float)mol_counts[DNA_G] / (float)nmol;

    gtree->lfreq[FREQ_A] = logf(freqa);
    gtree->lfreq[FREQ_T] = logf(freqt);
    gtree->lfreq[FREQ_C] = logf(freqc);
    gtree->lfreq[FREQ_G] = logf(freqg);

    /************************************************************
     * The following code adapted from LAMARC, (c) 2002
     * Peter Beerli, Mary Kuhner, Jon  Yamato and Joseph Felsenstein
     * TODO: license ref?
     ***********************************************************/
    pur = freqa + freqg;
    pyr = freqc + freqt;
    if (!pur || !pyr) {
        // TOO: handle error
    }
    freqar = freqa / pur;
    freqgr = freqg / pur;
    freqcy = freqc / pyr;
    freqty = freqt / pyr;
    gtree->lfreq[FREQ_AR] = log(freqar);
    gtree->lfreq[FREQ_GR] = log(freqgr);
    gtree->lfreq[FREQ_CY] = log(freqcy);
    gtree->lfreq[FREQ_TY] = log(freqty);
    ag = freqa * freqg;
    ct = freqc * freqt;
    m = (2.0 * pur * pyr) - (ag + ct);
    n = (ag / pur) + (ct / pyr);
    gtree->yrate = m / (m + n);
    gtree->xrate = 1.0 - gtree->yrate;
    fracchange = gtree->yrate * (2.0 * freqa * freqgr + 2.0 * freqc * freqty) +
                 gtree->xrate * (1.0 - freqa * freqa - freqc * freqc -
                                 freqg * freqg - freqt * freqt);
    gtree->xrate /= -(fracchange);
    gtree->yrate /= -(fracchange);

    /***********************************************************/

    //TODO: error checking (for this and other CUDA calls.)

    //These precomputed values are useful for likelihood calculation, and will never change
    //over the life of the program. Put them in constant memory for speed of access.
    hipMemcpyToSymbol(HIP_SYMBOL(lfreq), gtree->lfreq, MPCGS_NUM_FREQ_TERM * sizeof(float));

}

__device__ static void gnode_connect(struct gene_node *child, struct gene_node *parent)
{

    if (parent) {
        if (!parent->child1) {
            parent->child1 = child;
        } else if (!parent->child2) {
            parent->child2 = child;
        }
    }

    if (child) {
    	child->parent = parent;
    }
}

__device__ static void gnode_disconnect(struct gene_node *gnode)
{

    struct gene_node *parent;

    if (!gnode) {
        // TODO: handle error
    }

    parent = gnode->parent;

    if (parent) {
        if (parent->child1 == gnode) {
            parent->child1 = parent->child2;
            parent->child2 = NULL;
        } else {
            parent->child2 = NULL;
        }
    }

    gnode->parent = NULL;
}

__device__ static void gnode_extract(struct gene_node *gnode)
{

    if (!gnode) {
        // TODO handle error
    }

    if (gnode->tree->root == gnode) {
        gnode->tree->root = gnode->next;
    }

    if (gnode->tree->last == gnode) {
        gnode->tree->last = gnode->prev;
    }

    if (gnode->prev) {
        gnode->prev->next = gnode->next;
    }
    if (gnode->next) {
        gnode->next->prev = gnode->prev;
    }

    gnode->prev = NULL;
    gnode->next = NULL;
}

__device__ static void gnode_insert_after(struct gene_node *gnode, struct gene_node *prev)
{

    if (!gnode) {
        // TODO: handle error
    }

    gnode->prev = prev;
    if (prev) {
        gnode->next = prev->next;
        prev->next = gnode;
    } else {
        gnode->next = gnode->tree->root;
        gnode->tree->root = gnode;
    }

    if (gnode->next) {
        gnode->next->prev = gnode;
    } else {
        gnode->tree->last = gnode;
    }
}

void gtree_nodes_init_gpu(struct gene_tree *gtree, size_t ntips, size_t seq_len)
{

    struct gene_node *nodes;
    struct gene_node *prev = NULL;
    size_t num_nodes, nodesSz;
    int i;

    gtree->ntips = ntips;
    gtree->nnodes = ntips - 1;

    num_nodes = gtree->nnodes + gtree->ntips;
    nodesSz = num_nodes * sizeof(*nodes);
    hipMallocManaged(&nodes, nodesSz);
    memset(nodes, 0, nodesSz);

    //Precompute some likelihood kernel sizing parameters
    gtree->block_size = 2 * DEF_BLKSZ;
    do {
    	gtree->block_size /= 2;
    	gtree->shared_size = (gtree->block_size * num_nodes * NUM_BASE * sizeof(float)) +
    			ceil(((float)gtree->block_size / (float)WARPSZ) * sizeof(float));
    }while(gtree->shared_size > MAX_SHARED_SZ);

    gtree->num_blocks = ceil((float)seq_len/(float)gtree->block_size);

    hipMallocManaged(&gtree->block_scratch, gtree->num_blocks * sizeof(float));
    hipMalloc((void **)&gtree->node_list_scratch, num_nodes * sizeof(*gtree->node_list_scratch));
    hipMalloc((void **)&gtree->rand_scratch, (num_nodes + 1) * sizeof(*gtree->rand_scratch));
    hipDeviceSynchronize();


    for (i = 0; i < gtree->nnodes; i++) {
    	nodes[i].order = nodes[i].idx = i;
    	nodes[i].prev = prev;
    	nodes[i].tree = gtree;
    	if (prev) {
            prev->next = &nodes[i];
        }
        gtree->last = prev = &nodes[i];
    }

    for (i = gtree->nnodes; i < num_nodes; i++) {
        nodes[i].order = -1;
        nodes[i].idx = i;
        nodes[i].tree = gtree;
    }

    gtree->nodes = nodes;
    gtree->tips = &nodes[gtree->nnodes];
    gtree->root = &nodes[0];

}

__device__ static void gtree_copy(struct gene_tree *gtree, struct gene_tree *newtree)
{

    float *block_scratch;
    struct gene_node **node_list_scratch;
    struct gene_node *newnodes;
    struct gene_node *gnode;
    size_t nodesSz;
    int i;

    if (!gtree || !newtree) {
        // TODO: handle error
    }

    nodesSz = sizeof(*gtree->nodes) * (gtree->nnodes + gtree->ntips);

    newnodes = newtree->nodes;
    memcpy(newnodes, gtree->nodes, nodesSz);

    block_scratch = newtree->block_scratch;
    node_list_scratch = newtree->node_list_scratch;
    *newtree = *gtree;
    newtree->block_scratch = block_scratch;
    newtree->node_list_scratch = node_list_scratch;

    for (i = 0; i < newtree->nnodes + newtree->ntips; i++) {
        // TODO: find a better way to do this
        gnode = &newnodes[i];
        if (gnode->parent) {
            gnode->parent = &newnodes[gnode->parent - gtree->nodes];
        }
        if (gnode->child1) {
            gnode->child1 = &newnodes[gnode->child1 - gtree->nodes];
        }
        if (gnode->child2) {
            gnode->child2 = &newnodes[gnode->child2 - gtree->nodes];
        }
        if (gnode->prev) {
            gnode->prev = &newnodes[gnode->prev - gtree->nodes];
        }
        if (gnode->next) {
            gnode->next = &newnodes[gnode->next - gtree->nodes];
        }
        gnode->tree = newtree;
        if (gnode->order == 0) {
            newtree->root = gnode;
        }
        if (gnode->order == (newtree->nnodes - 1)) {
            newtree->last = gnode;
        }
    }

    newtree->nodes = newnodes;
    newtree->tips = &newnodes[gtree->nnodes];
}

__device__ static void gtree_fixup_order(struct gene_tree *gtree, struct gene_node *stopat)
{

    struct gene_node *node;

    if (!gtree) {
        // TODO: handle error
    }

    gtree->root->order = 0;
    for (node = gtree->root; node != gtree->last; node = node->next) {
        if (node == stopat) {
            break;
        }
        node->next->order = node->order + 1;
    }
}

__device__ static void gnode_list_init(size_t list_size, struct gnode_list *list)
{

	int i;

	list->head = 0;
    list->tail = 0;

    for(i = 0; i < list_size; i++) {
    	list->gnodes[i] = NULL;
    }

}


__device__ static void gnode_list_destroy(struct gnode_list *list)
{

    free(list->gnodes);
}

__device__ static unsigned int gnode_list_get_size(struct gnode_list *list)
{

    if (!list) {
        // TODO: warn
        return (0);
    }

    return (list->head - list->tail);
}

__device__ static void gnode_list_enqueue(struct gnode_list *list, struct gene_node *node)
{

    if (!list) {
        // TODO: handle error
    }

    list->gnodes[list->head++] = node;
}

__device__ static void gnode_list_collate_head(struct gnode_list *list)
{

    struct gene_node *collate_target;
    struct gene_node **target_pos;
    float parent_time;

    if (!list) {
        // TODO: handle error
    }

    if (gnode_list_get_size(list) <= 1) {
        return;
    }

    collate_target = list->gnodes[list->head - 1];
    if (collate_target->parent) {
        parent_time = collate_target->parent->time;
    } else {
        parent_time = FLT_MAX;
    }

    target_pos = &list->gnodes[list->head - 1];
    while (target_pos > &list->gnodes[list->tail]) {
        *target_pos = *(target_pos - 1);
        if (parent_time > (*target_pos)->parent->time) {
            break;
        }
        target_pos--;
    }

    *target_pos = collate_target;
}

__device__ static struct gene_node *gnode_list_dequeue(struct gnode_list *list)
{

    struct gene_node *node;

    if (!list) {
        // TODO: handle error
    }

    if (list->head == list->tail) {
        return (NULL);
    }

    node = list->gnodes[list->tail++];

    return (node);
}

__device__ static struct gene_node *gnode_list_get_tail(struct gnode_list *list)
{

    if (!list) {
        // TODO: handle error
    }

    return (list->gnodes[list->tail]);
}

__device__ static int gnode_list_empty(struct gnode_list *list)
{

    if (!list) {
        // TODO: handle error
    }

    return (list->head == list->tail);
}

__device__ static void gnode_set_exp_gpu(struct gene_node *gnode, float xrate, float yrate, unsigned set_children)
{

	struct gene_node *parent;
	float length, n1, n2;

	if (!gnode) {
		// TODO: handle error
	}

	parent = gnode->parent;
	if (parent && !gnode->exp_valid) {
		length = parent->time - gnode->time;
		/********************************************************
		 * The following code adapted from LAMARC, (c) 2002
		 * Peter Beerli, Mary Kuhner, Jon  Yamato and Joseph Felsenstein
		 * TODO: license ref?
		 ********************************************************/
		n1 = (length * xrate) / 2.0;
		n2 = (length * yrate) / 2.0;
		gnode->lexpA = n1 + log(exp(-n1) - exp(n1));
		gnode->lexpB = (2.0 * n1) + (length * yrate);
		gnode->lexpC = (2.0 * n1) + n2 + log(exp(-n2) - exp(n2));
		/*******************************************************/
		gnode->exp_valid = 1;
	}
	if(set_children) {
		if (gnode->child1) {
			gnode_set_exp_gpu(gnode->child1, xrate, yrate, 0);
		}
		if (gnode->child2) {
			gnode_set_exp_gpu(gnode->child2, xrate, yrate, 0);
		}
	}

}

__device__ static struct gene_node *gnode_list_get_random_gpu (struct gnode_list *list,
                                               float rand_float)
{

	unsigned list_pos, list_size;

    if (!list) {
        // TODO: handle error
    }

    if (gnode_list_get_size(list) == 0) {
        return (NULL);
    }

    list_size = gnode_list_get_size(list);
    list_pos = rand_float * list_size;
    if(list_pos >= list_size) {
    	list_pos = list_size - 1;
    }

    return (list->gnodes[list_pos]);
}

__device__ static float get_next_coal_time_gpu(unsigned int active,
                         unsigned int inactive,
                         unsigned int act_coal,
                         float theta,
                         float rand_float)
{

    float r, time, denom;


    if (active < act_coal || inactive < 2 - act_coal) {
        return (FLT_MAX);
    }

    r = rand_float;

    if (2 == act_coal) {
        denom = ((float)active * (float)(active - 1)) / 2.0;
    } else if (1 == act_coal) {
        denom = (float)active * (float)inactive;
    } else {
        return (FLT_MAX);
    }

    time = -(log(r) * theta) / (2.0 * denom);

    return (time);
}

__device__ void fill_rand_array(float *rand_array, unsigned count, hiprandStateMtgp32 *mtgp)
{

	int i;

	for(i = 0; i < count; i++) {
		rand_array[i] = hiprand_uniform(mtgp);
		if(rand_array[i] == 0) {
			rand_array[i] = FLT_MIN;
		}
	}

}

__device__ void gtree_propose_fixed_target_gpu(struct gene_tree *current,
                                             struct gene_tree *proposal,
                                             float theta,
                                             unsigned int tgtidx)
{

    struct gene_node *target, *parent, *gparent, *newgparent, *node, *tail;
    struct gene_node *child1, *child2, *oldsibling, *sibling, *newnode;
    struct gene_node *ival_end;
    struct gnode_list ival_list;
    float currT, nextT, eventT;
    float *rand_scratch;
    float rand_float;

    if (!current) {
        // TODO: handle error
    }

    if (tgtidx >= (current->nnodes + current->ntips) ||
        tgtidx == proposal->root->idx) {
        // TODO: handle error
    }

    ival_list.gnodes = proposal->node_list_scratch;

    gnode_list_init((current->nnodes + current->ntips), &ival_list);

    gtree_copy(current, proposal);

    target = &proposal->nodes[tgtidx];
    if (target >= proposal->root) {
        target++;
    }
    parent = target->parent;

    gnode_disconnect(target);
    oldsibling = parent->child1;

    if (target->time == 0) {
        // target is a tip
        node = proposal->last;
    } else {
        node = target->prev;
    }
    while (node) {
        child1 = node->child1;
        child2 = node->child2;
        if (child1 && child1->time <= target->time) {
            gnode_list_enqueue(&ival_list, child1);
        }
        if (child2 && child2->time <= target->time) {
            gnode_list_enqueue(&ival_list, child2);
        }
        node = node->prev;
    }

    rand_scratch = proposal->rand_scratch;

    /********************************************************
     * The following code adapted from LAMARC, (c) 2002
     * Peter Beerli, Mary Kuhner, Jon  Yamato and Joseph Felsenstein
     * TODO: license ref?
     ********************************************************/
    currT = target->time;

    while (1) {
        if (gnode_list_empty(&ival_list)) {
            // TODO: handle error
        }

        ival_end = (gnode_list_get_tail(&ival_list))->parent;
        rand_float = *rand_scratch;
        rand_scratch++;
        if (ival_end) {
            nextT = ival_end->time;
            eventT = get_next_coal_time_gpu(
              1, gnode_list_get_size(&ival_list), 1, theta, rand_float);
        } else {
            nextT = FLT_MAX;
            eventT = get_next_coal_time_gpu(2, 0, 2, theta, rand_float);
        }
        if ((currT + eventT) < nextT) {

        	rand_float = *rand_scratch;
        	rand_scratch++;
            sibling = gnode_list_get_random_gpu(&ival_list, rand_float);
            if (sibling == parent) {
                // Parent is a stick at this point, so it can't be a sibling.
                sibling = parent->child1;
            }

            newnode = parent; // for clarity
            gparent = parent->parent;
            newgparent = sibling->parent;

            if (parent != ival_end) {
                gnode_extract(parent);
                gnode_insert_after(newnode, ival_end);
            }
            if (parent != sibling->parent) {
                gnode_disconnect(oldsibling);
                gnode_disconnect(parent);
                gnode_disconnect(sibling);
                gnode_connect(oldsibling, gparent);
                gnode_connect(newnode, newgparent);
                gnode_connect(sibling, newnode);
            }

            gnode_connect(target, newnode);
            newnode->time = currT + eventT;
            newnode->exp_valid = 0;
            target->exp_valid = 0;
            sibling->exp_valid = 0;

            gtree_fixup_order(proposal, target);
            gnode_set_exp_gpu(newnode, proposal->xrate, proposal->yrate, 1);

            break;

        } else {
            node = gnode_list_dequeue(&ival_list);
            if (!gnode_list_empty(&ival_list)) {
                tail = gnode_list_get_tail(&ival_list);
                if (tail->parent == node->parent) {
                    // TODO: some explanation here
                    gnode_list_dequeue(&ival_list);
                }
            }
            // parent is guaranteed to exist since we would have merged the root
            // otherwise.
            gnode_list_enqueue(&ival_list, node->parent);
            gnode_list_collate_head(&ival_list);
        }

        currT = nextT;
    }

    /***********************************************************/

    gnode_list_destroy(&ival_list);

}

#endif /* MPCGS_NOGPU */
}
